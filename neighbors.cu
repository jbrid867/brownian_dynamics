#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <time.h>

using namespace std;

#define BLOCK_SIZE 256

/*__global__ void NN_make(float * NNmat, float coords_list, int numC) 
{
	// for one crowder, do all numC-1 NNs
}

__global__ void start_sys(float * coords, int numC)
{

}*/

__global__ void init_RNG(hiprandState *rngStates,
							  const unsigned int seed, int xLen)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tid = Row*xLen + Col;

	hiprand_init(seed+tid, tid, 0, &rngStates[tid]);
}

__global__ void rand_gen(hiprandState *state, float * rands, int xLen, int yLen)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int idx=Row*xLen + Col;
	if(idx<yLen*xLen)
		rands[idx] = hiprand_uniform(&state[idx]);
}

void wrapper(int switcher)
{
	int seed=(int)time(NULL), xLen=100, yLen=10;
	hiprandState *d_state;
	hiprandState *state;
	float *rands;
	float *d_rands;

	rands=(float *)malloc(xLen*yLen*sizeof(float));
	state=(hiprandState *)malloc(xLen*yLen*sizeof(hiprandState));


	hipMalloc((void **)&d_state, xLen*yLen*sizeof(hiprandState));
	hipMalloc((void **)&d_rands, xLen*yLen*sizeof(float));

	hipMemcpy(d_rands, rands, xLen*yLen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_state, state, xLen*yLen*sizeof(hiprandState), hipMemcpyHostToDevice);



	dim3 Grid((xLen-1)/32 + 1, (yLen-1)/32 +1);
	dim3 Blocks(32,32) ;
	init_RNG<<<Grid,Blocks>>>(d_state, seed, xLen);
	hipMemcpy(state, d_state, xLen*yLen*sizeof(hiprandState), hipMemcpyDeviceToHost);
	rand_gen<<<Grid,Blocks>>>(d_state, d_rands, xLen, yLen);
	hipMemcpy(rands, d_rands, xLen*yLen*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i<xLen*yLen;i+=10)
		printf("random number %d is %f.\n",i,rands[i]);

	hipFree(d_state);
	hipFree(d_rands);
	free(state);
	free(rands);
}

int main()
{
	wrapper(0);
	return 0;
}
/*void gpu_NN_start(float coords_list, float * NNlist, int numC) // wrapper to initiate kernels
{
	// coords_list needs to be 1D list of coords
	// allocate host and device memory
	// copy coords list to gpu
	// set up grid and blocks
	// execute kernel
	// copy nnlist back
	// free memory
	// exit
}*/