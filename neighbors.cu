#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <time.h>
//#include <vector.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define BLOCK_SIZE 256

/*__global__ void NN_make(float * NNmat, float coords_list, int numC) 
{
	// for one crowder, do all numC-1 NNs
}

__global__ void start_sys(float * coords, int numC)
{

}*/

/*__global__ void init_RNG(curandState *rngStates,
							  const unsigned int seed, int xLen)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tid = Row*xLen + Col;

	curand_init(seed+tid, tid, 0, &rngStates[tid]);
}

__global__ void rand_gen(curandState *state, float * rands, int xLen, int yLen)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int idx=Row*xLen + Col;
	if(idx<yLen*xLen)
		rands[idx] = curand_normal(&state[idx]);
}*/

/*__global__ void coord_ICs(float *coords, float params[])
{
	float space=params[0], Len=params[1];
	int N=params[2], n=params[3];
	int end=params[4]; // gives the index of the last element of params
	int indices[3];
	indices[0]=blockIdx.x*blockDim.x + threadIdx.x ;
	indices[1]=blockIdx.y*blockDim.y + threadIdx.y ;
	indices[2]=blockIdx.z*blockDim.z + threadIdx.z ;
	int arrIdx=(indices[0] + indices[1]*n + indices[2]*n*n)*3;
	if(indices[0]<n && indices[1]<n && indices[2]<n)
		for(int i=0;i<3;i++)
		{
			coords[arrIdx+i]=(indices[i]+0.5)*space - Len;
		}

}*/

void crowd_build_wrap(float *coords, float params[])
{/*
	// params has spacing, total length, N, n = (N)^1/3
	//N needs to not include the subtracted points yet
	float *d_coords;
	float space=params[0], Len=params[1];
	int N=params[2], n=params[3];
	int end=params[4]; // gives the index of the last element of params
	//int N_red=0; // reduced number of proteins
	

	cudaMalloc((void **) &d_coords, N*3*sizeof(float));

	dim3 Grid((n-1)/4 +1, (n-1)/4 +1, (n-1)/4 + 1);
	dim3 Blocks(8,8,8);

	coord_ICs<<<Grid, Blocks>>>(d_coords, n, space, Len);
	gpuErrchk( cudaPeekAtLastError() );
	gpuErrchk(cudaDeviceSynchronize());
	gpuErrchk(cudaMemcpy(coords, d_coords, N*3*sizeof(float), cudaMemcpyDeviceToHost));
	cudaFree(d_coords);
	printf("c=%f\n", coords[1]);*/
}

void rand_wrapper(int switcher)
{
	/*int seed=(int)time(NULL), xLen=100, yLen=10;
	curandState *d_state;
	curandState *state;
	float *rands;
	float *d_rands;

	rands=(float *)malloc(xLen*yLen*sizeof(float));
	state=(curandState *)malloc(xLen*yLen*sizeof(curandState));


	cudaMalloc((void **)&d_state, xLen*yLen*sizeof(curandState));
	cudaMalloc((void **)&d_rands, xLen*yLen*sizeof(float));

	cudaMemcpy(d_rands, rands, xLen*yLen*sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(d_state, state, xLen*yLen*sizeof(curandState), cudaMemcpyHostToDevice);



	dim3 Grid((xLen-1)/32 + 1, (yLen-1)/32 +1);
	dim3 Blocks(32,32) ;
	init_RNG<<<Grid,Blocks>>>(d_state, seed, xLen);
	cudaMemcpy(state, d_state, xLen*yLen*sizeof(curandState), cudaMemcpyDeviceToHost);
	rand_gen<<<Grid,Blocks>>>(d_state, d_rands, xLen, yLen);
	cudaMemcpy(rands, d_rands, xLen*yLen*sizeof(float), cudaMemcpyDeviceToHost);
	for(int i=0;i<xLen*yLen;i+=10)
		printf("random number %d is %f.\n",i,rands[i]);

	cudaFree(d_state);
	cudaFree(d_rands);
	free(state);
	free(rands);*/
}



/*int main()
{
	int N=10;
	int n=ceil(pow(N,0.33333));
	double L=10.0;
	double l= (double)2*L/n;
	float in[4]; // 0: space, 1: length, 2: N, 3: n
	in[0]=l;
	in[1]=L;
	in[2]=n*n*n;
	in[3]=n;
	float coords[n*n*n*3];
	for(int i=0;i<n*n*n*3;i++)
		coords[i]=0;
	crowd_build_wrap(coords, in);
	for(int i=0;i<n*n*n*3;i++)
		printf("c = %f\n",coords[i] );
	//rand_wrapper(0);
	//rand_wrapper(0);
	return 0;
}*/

/*void gpu_NN_start(float coords_list, float * NNlist, int numC) // wrapper to initiate kernels
{
	// coords_list needs to be 1D list of coords
	// allocate host and device memory
	// copy coords list to gpu
	// set up grid and blocks
	// execute kernel
	// copy nnlist back
	// free memory
	// exit
}*/