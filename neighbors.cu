#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <time.h>
//#include <vector.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define BLOCK_SIZE 256

/*__global__ void NN_make(float * NNmat, float coords_list, int numC) 
{
	// for one crowder, do all numC-1 NNs
}

__global__ void start_sys(float * coords, int numC)
{

}*/



/*__global__ void coord_ICs(float *coords, float params[])
{
	float space=params[0], Len=params[1];
	int N=params[2], n=params[3];
	int end=params[4]; // gives the index of the last element of params
	int indices[3];
	indices[0]=blockIdx.x*blockDim.x + threadIdx.x ;
	indices[1]=blockIdx.y*blockDim.y + threadIdx.y ;
	indices[2]=blockIdx.z*blockDim.z + threadIdx.z ;
	int arrIdx=(indices[0] + indices[1]*n + indices[2]*n*n)*3;
	if(indices[0]<n && indices[1]<n && indices[2]<n)
		for(int i=0;i<3;i++)
		{
			coords[arrIdx+i]=(indices[i]+0.5)*space - Len;
		}

}*/

__global__ void NN_lists(float *coords, int *NNs, int N, int n, float L)
{
	int xdx, ydx, count;
	xdx=blockIdx.x*blockDim.x + threadIdx.x;
	ydx=blockIdx.y*blockDim.y + threadIdx.y;
	float x1=0, y1=0, z1=0, x2=0, y2=0, z2=0,dx,dy,dz;
	float mag2;
	int index=xdx + n*ydx; // n should be sqrt(N)
	//float cut=2*L/n; // the lattice spacing
	float mag2s[10];
	float difference, difference2;
	int remove_index;
	bool remove_bool=false, first;
	
	__shared__ int coords_mz[300];
	__shared__ int coords_my[300];
	__shared__ int coords_mx[300];
	__shared__ int coords_pz[300];
	__shared__ int coords_py[300];
	__shared__ int coords_px[300];
	if(index<300){
		coords_mx[index]=coords[index*3]-2*L;
		coords_my[index]=coords[index*3+1]-2*L;
		coords_mz[index]=coords[index*3+2]-2*L;
		coords_px[index]=coords[index*3]+2*L;
		coords_py[index]=coords[index*3+1]+2*L;
		coords_pz[index]=coords[index*3+2]+2*L;
	

		__syncthreads();
		

		x1=coords[3*index];
		y1=coords[3*index+1];
		z1=coords[3*index+2];

		count=0;
		for(int i=0; i<N; i++)
		{	
			first=true;
			if(i!=index)
			{
				x2=coords[3*i];
				y2=coords[3*i+1];
				z2=coords[3*i+2];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;
				if(count<10)
				{
					NNs[10*index+count]=i;
					mag2s[count]=mag2;
					count++;
				}
				else
				{	
					
					remove_bool=false;
					first=true;
					for(int j=0;j<10;j++)
					{
						if(mag2<mag2s[j])
						{	
							difference=mag2s[j]-mag2;
							remove_bool=true;
							if(first)
							{
								remove_index=j;
								difference2=difference;
								first=false;
							}
							else if(difference<difference2)
							{
								remove_index=j;
								difference2=difference;
							}
						}
					}
					if(remove_bool)
					{
						NNs[10*index+remove_index]=i;
						mag2s[remove_index]=mag2;
					}
				}
			}
		}

		for(int i=0;i<N;i++) // check for neigbors through the boundaries. 
		{
			//-x
			x2=coords_mx[i];
			y2=coords[3*i+1];
			z2=coords[3*i+2];
			dx=x2-x1;dy=y2-y1;dz=z2-z1;
			mag2=dx*dx+dy*dy+dz*dz;
			if(i!=index)
			{
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+N;
					mag2s[remove_index]=mag2;
				}
				//-y////////////////////////////////////////////////////////////////
				////////////////////////////////////////////////////////////////////
				y2=coords_my[i];
				x2=coords[3*i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+2*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-z
				////////////////////////////////////////////////////////////////
				z2=coords_mz[i];
				y2=coords[3*i+1];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+3*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x
				////////////////////////////////////////////////////////////////
				z2=coords[3*i+2];
				x2=coords_px[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+4*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+y
				////////////////////////////////////////////////////////////////
				y2=coords_py[i];
				x2=coords[3*i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+5*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+z
				////////////////////////////////////////////////////////////////
				y2=coords[3*i+1];
				z2=coords_pz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+6*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x + y
				////////////////////////////////////////////////////////////////
				y2=coords_py[i];
				x2=coords_px[i];
				z2=coords[3*i+2];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+7*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x -y
				////////////////////////////////////////////////////////////////
				y2=coords_my[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+8*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x +y
				////////////////////////////////////////////////////////////////
				y2=coords_py[i];
				x2=coords_mx[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+9*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x -y
				////////////////////////////////////////////////////////////////
				y2=coords_my[i];
				x2=coords_mx[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+10*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x +z
				////////////////////////////////////////////////////////////////
				y2=coords[3*i+1];
				x2=coords_px[i];
				z2=coords_pz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+11*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x -z
				////////////////////////////////////////////////////////////////
				z2=coords_mz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+12*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x +z
				////////////////////////////////////////////////////////////////
				x2=coords_mx[i];
				z2=coords_pz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+13*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x -z
				////////////////////////////////////////////////////////////////
				x2=coords_mx[i];
				z2=coords_mz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+14*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+y +z
				////////////////////////////////////////////////////////////////
				y2=coords_py[i];
				x2=coords[3*i];
				z2=coords_pz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+15*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+y -z
				////////////////////////////////////////////////////////////////
				z2=coords_mz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+16*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-y +z
				////////////////////////////////////////////////////////////////
				y2=coords_my[i];
				z2=coords_pz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+17*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-y -z
				////////////////////////////////////////////////////////////////
				z2=coords_mz[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+18*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x+y+z
				////////////////////////////////////////////////////////////////
				z2=coords_pz[i];
				y2=coords_py[i];
				x2=coords_px[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+19*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x-y-z
				////////////////////////////////////////////////////////////////
				z2=coords_mz[i];
				y2=coords_my[i];
				x2=coords_mx[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+20*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x-y-z
				////////////////////////////////////////////////////////////////
				x2=coords_px[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+21*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x+y-z
				////////////////////////////////////////////////////////////////
				x2=coords_mx[i];
				y2=coords_py[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+22*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x-y+z
				////////////////////////////////////////////////////////////////
				z2=coords_pz[i];
				y2=coords_my[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+23*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x+y-z
				////////////////////////////////////////////////////////////////
				z2=coords_mz[i];
				y2=coords_py[i];
				x2=coords_px[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+24*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//+x-y+z
				////////////////////////////////////////////////////////////////
				z2=coords_pz[i];
				y2=coords_my[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+25*N;
					mag2s[remove_index]=mag2;
				}
				////////////////////////////////////////////////////////////////
				//-x+y+z
				////////////////////////////////////////////////////////////////
				x2=coords_mx[i];
				y2=coords_py[i];
				dx=x2-x1;dy=y2-y1;dz=z2-z1;
				mag2=dx*dx+dy*dy+dz*dz;		
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i+26*N;
					mag2s[remove_index]=mag2;
				}
			}			
		}
	}
}


void make_NNs(float *coords, int *NN, float params[])
{
	printf("how about here????");
	float space=params[0];
	float length=params[1];
	int N=params[2], n=params[3];

	float *device_coords;
	int *device_NNs; // store X neighbors per crowder?
	//int *system_NNs;
	int gridsize=ceil(pow(N,0.5));
	hipMalloc((void **) &device_coords, N*3*sizeof(float));
	hipMalloc((void **) &device_NNs, N*10*sizeof(int));

	hipMemcpy(device_coords, coords, N*3*sizeof(float), hipMemcpyHostToDevice);
	//system_NNs=(int *)malloc(10*N*sizeof(int));

	dim3 Grid(gridsize, gridsize, 1);
	dim3 Blocks(16,16);

	NN_lists<<<1, N>>>(device_coords, device_NNs, N, n, length);
	hipMemcpy(NN, device_NNs, N*10*sizeof(int), hipMemcpyDeviceToHost);

	printf("Crowder #1's nearest neighbors are\n");
	
	for(int i=0;i<10;i++)
	{
		printf("%i\n", NN[i]);
	}	
	hipFree(device_coords);
	hipFree(device_NNs);
}



void crowd_build_wrap(float *coords, float params[])
{/*
	// params has spacing, total length, N, n = (N)^1/3
	//N needs to not include the subtracted points yet
	float *d_coords;
	float space=params[0], Len=params[1];
	int N=params[2], n=params[3];
	int end=params[4]; // gives the index of the last element of params
	//int N_red=0; // reduced number of proteins
	

	cudaMalloc((void **) &d_coords, N*3*sizeof(float));

	dim3 Grid((n-1)/4 +1, (n-1)/4 +1, (n-1)/4 + 1);
	dim3 Blocks(8,8,8);

	coord_ICs<<<Grid, Blocks>>>(d_coords, n, space, Len);
	gpuErrchk( cudaPeekAtLastError() );
	gpuErrchk(cudaDeviceSynchronize());
	gpuErrchk(cudaMemcpy(coords, d_coords, N*3*sizeof(float), cudaMemcpyDeviceToHost));
	C
	printf("c=%f\n", coords[1]);*/
}

__global__ void init_RNG(hiprandState *rngStates,
							  const unsigned int seed, int n)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tid = Row*n + Col;

	hiprand_init(seed+tid, tid, 0, &rngStates[tid]);
}

__global__ void rand_gen(hiprandState *state, float * rands, int n, float sigma)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int idx=Row*n + Col;
	float x,y,z,mag2=0,mag, random;
	if(idx<n*n)
	{
		random = hiprand_normal(&state[idx]);
		random=random*sigma;
		x=hiprand_uniform(&state[idx]);
		y=hiprand_uniform(&state[idx]);
		z=hiprand_uniform(&state[idx]);
		mag2=x*x+y*y+z*z;
		mag=sqrt(mag2);
		x=x/mag;y=y/mag;z=z/mag;
		rands[3*idx]=x*random;
		rands[3*idx+1]=y*random;
		rands[3*idx+2]=z*random;
	}
}

void rand_wrapper(int num, float *steparr, float sigma)
{
	int seed=(int)time(NULL);
	hiprandState *d_state;
	hiprandState *state;
	int n = ceil(pow(num,0.5));
	//float *rands;
	float *d_steps;

	//rands=(float *)malloc(xLen*yLen*sizeof(float));
	state=(hiprandState *)malloc(num*sizeof(hiprandState));


	hipMalloc((void **)&d_state, num*sizeof(hiprandState));
	hipMalloc((void **)&d_steps, 3*num*sizeof(float));

	hipMemcpy(d_steps, steparr, 3*num*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_state, state, num*sizeof(hiprandState), hipMemcpyHostToDevice);



	dim3 Grid((n/32) + 1, (n/32) + 1);
	dim3 Blocks(32,32) ;
	init_RNG<<<Grid,Blocks>>>(d_state, seed, n);
	hipMemcpy(state, d_state, num*sizeof(hiprandState), hipMemcpyDeviceToHost);
	rand_gen<<<Grid,Blocks>>>(d_state, d_steps, n, sigma);
	hipMemcpy(steparr, d_steps, 3*num*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i<3*num;i+=10)
		printf("random number %d is %f.\n",i,steparr[i]);

	hipFree(d_state);
	hipFree(d_steps);
	free(state);
}



/*int main()
{
	int N=10;
	int n=ceil(pow(N,0.33333));
	double L=10.0;
	double l= (double)2*L/n;
	float in[4]; // 0: space, 1: length, 2: N, 3: n
	in[0]=l;
	in[1]=L;
	in[2]=n*n*n;
	in[3]=n;
	float coords[n*n*n*3];
	for(int i=0;i<n*n*n*3;i++)
		coords[i]=0;
	crowd_build_wrap(coords, in);
	for(int i=0;i<n*n*n*3;i++)
		printf("c = %f\n",coords[i] );
	//rand_wrapper(0);
	//rand_wrapper(0);
	return 0;
}*/

/*void gpu_NN_start(float coords_list, float * NNlist, int numC) // wrapper to initiate kernels
{
	// coords_list needs to be 1D list of coords
	// allocate host and device memory
	// copy coords list to gpu
	// set up grid and blocks
	// execute kernel
	// copy nnlist back
	// free memory
	// exit
}*/