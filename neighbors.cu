#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath.h>

using namespace std;

#define BLOCK_SIZE 256

__global__ void NN_make(float * NNmat, float coords_list, int numC) 
{
	// for one crowder, do all numC-1 NNs
}

void gpu_NN_start(float coords_list, float * NNlist, int numC) // wrapper to initiate kernels
{
	// coords_list needs to be 1D list of coords
	// allocate host and device memory
	// copy coords list to gpu
	// set up grid and blocks
	// execute kernel
	// copy nnlist back
	// free memory
	// exit
}