#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>
#include <time.h>
//#include <vector.h>

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define BLOCK_SIZE 256

/*__global__ void NN_make(float * NNmat, float coords_list, int numC) 
{
	// for one crowder, do all numC-1 NNs
}

__global__ void start_sys(float * coords, int numC)
{

}*/

/*__global__ void init_RNG(curandState *rngStates,
							  const unsigned int seed, int xLen)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tid = Row*xLen + Col;

	curand_init(seed+tid, tid, 0, &rngStates[tid]);
}

__global__ void rand_gen(curandState *state, float * rands, int xLen, int yLen)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int idx=Row*xLen + Col;
	if(idx<yLen*xLen)
		rands[idx] = curand_normal(&state[idx]);
}*/

/*__global__ void coord_ICs(float *coords, float params[])
{
	float space=params[0], Len=params[1];
	int N=params[2], n=params[3];
	int end=params[4]; // gives the index of the last element of params
	int indices[3];
	indices[0]=blockIdx.x*blockDim.x + threadIdx.x ;
	indices[1]=blockIdx.y*blockDim.y + threadIdx.y ;
	indices[2]=blockIdx.z*blockDim.z + threadIdx.z ;
	int arrIdx=(indices[0] + indices[1]*n + indices[2]*n*n)*3;
	if(indices[0]<n && indices[1]<n && indices[2]<n)
		for(int i=0;i<3;i++)
		{
			coords[arrIdx+i]=(indices[i]+0.5)*space - Len;
		}

}*/

__global__ void NN_lists(float *coords, int *NNs, int N, int n, float L)
{
	int xdx, ydx, count;
	xdx=blockIdx.x*blockDim.x + threadIdx.x;
	ydx=blockIdx.y*blockDim.y + threadIdx.y;
	float x1=0, y1=0, z1=0, x2=0, y2=0, z2=0,dx,dy,dz;
	float mag2;
	int index=xdx + n*ydx; // n should be sqrt(N)
	//float cut=2*L/n; // the lattice spacing
	float mag2s[10];
	float difference, difference2;
	int remove_index;
	bool remove_bool=false, first;
	

	x1=coords[3*index];
	y1=coords[3*index+1];
	z1=coords[3*index+2];

	count=0;
	for(int i=0; i<N; i++)
	{	
		first=true;
		if(i!=index)
		{
			x2=coords[3*i];
			y2=coords[3*i+1];
			z2=coords[3*i+2];
			dx=x2-x1;dy=y2-y1;dz=z2-z1;
			mag2=dx*dx+dy*dy+dz*dz;
			if(count<10)
			{
				NNs[10*index+count]=i;
				mag2s[count]=mag2;
				count++;
			}
			else
			{	
				
				remove_bool=false;
				first=true;
				for(int j=0;j<10;j++)
				{
					if(mag2<mag2s[j])
					{	
						difference=mag2s[j]-mag2;
						remove_bool=true;
						if(first)
						{
							remove_index=j;
							difference2=difference;
							first=false;
						}
						else if(difference<difference2)
						{
							remove_index=j;
							difference2=difference;
						}
					}
				}
				if(remove_bool)
				{
					NNs[10*index+remove_index]=i;
					mag2s[remove_index]=mag2;
				}
			}
		}
	}
}



void make_NNs(float *coords, float params[])
{
	printf("how about here????");
	float space=params[0];
	float length=params[1];
	int N=params[2], n=params[3];

	float *device_coords;
	int *device_NNs; // store X neighbors per crowder?
	int *system_NNs;
	int gridsize=ceil(pow(N,0.5));
	hipMalloc((void **) &device_coords, N*3*sizeof(float));
	hipMalloc((void **) &device_NNs, N*10*sizeof(int));

	hipMemcpy(device_coords, coords, N*3*sizeof(float), hipMemcpyHostToDevice);
	system_NNs=(int *)malloc(10*N*sizeof(int));

	dim3 Grid(gridsize, gridsize, 1);
	dim3 Blocks(16,16);

	NN_lists<<<Grid, Blocks>>>(device_coords, device_NNs, N, n, length);
	hipMemcpy(system_NNs, device_NNs, N*10*sizeof(int), hipMemcpyDeviceToHost);

	printf("Crowder #21's nearest neighbors are\n");
	for(int i=0;i<10;i++)
	{
		printf("%i\n", system_NNs[210+i]);
	}

	// cudaFree(device_coords);
	// cudaFree(device_NNs);
}



void crowd_build_wrap(float *coords, float params[])
{/*
	// params has spacing, total length, N, n = (N)^1/3
	//N needs to not include the subtracted points yet
	float *d_coords;
	float space=params[0], Len=params[1];
	int N=params[2], n=params[3];
	int end=params[4]; // gives the index of the last element of params
	//int N_red=0; // reduced number of proteins
	

	cudaMalloc((void **) &d_coords, N*3*sizeof(float));

	dim3 Grid((n-1)/4 +1, (n-1)/4 +1, (n-1)/4 + 1);
	dim3 Blocks(8,8,8);

	coord_ICs<<<Grid, Blocks>>>(d_coords, n, space, Len);
	gpuErrchk( cudaPeekAtLastError() );
	gpuErrchk(cudaDeviceSynchronize());
	gpuErrchk(cudaMemcpy(coords, d_coords, N*3*sizeof(float), cudaMemcpyDeviceToHost));
	C
	printf("c=%f\n", coords[1]);*/
}

void rand_wrapper(int switcher)
{
	/*int seed=(int)time(NULL), xLen=100, yLen=10;
	curandState *d_state;
	curandState *state;
	float *rands;
	float *d_rands;

	rands=(float *)malloc(xLen*yLen*sizeof(float));
	state=(curandState *)malloc(xLen*yLen*sizeof(curandState));


	cudaMalloc((void **)&d_state, xLen*yLen*sizeof(curandState));
	cudaMalloc((void **)&d_rands, xLen*yLen*sizeof(float));

	cudaMemcpy(d_rands, rands, xLen*yLen*sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(d_state, state, xLen*yLen*sizeof(curandState), cudaMemcpyHostToDevice);



	dim3 Grid((xLen-1)/32 + 1, (yLen-1)/32 +1);
	dim3 Blocks(32,32) ;
	init_RNG<<<Grid,Blocks>>>(d_state, seed, xLen);
	cudaMemcpy(state, d_state, xLen*yLen*sizeof(curandState), cudaMemcpyDeviceToHost);
	rand_gen<<<Grid,Blocks>>>(d_state, d_rands, xLen, yLen);
	cudaMemcpy(rands, d_rands, xLen*yLen*sizeof(float), cudaMemcpyDeviceToHost);
	for(int i=0;i<xLen*yLen;i+=10)
		printf("random number %d is %f.\n",i,rands[i]);

	cudaFree(d_state);
	cudaFree(d_rands);
	free(state);
	free(rands);*/
}



/*int main()
{
	int N=10;
	int n=ceil(pow(N,0.33333));
	double L=10.0;
	double l= (double)2*L/n;
	float in[4]; // 0: space, 1: length, 2: N, 3: n
	in[0]=l;
	in[1]=L;
	in[2]=n*n*n;
	in[3]=n;
	float coords[n*n*n*3];
	for(int i=0;i<n*n*n*3;i++)
		coords[i]=0;
	crowd_build_wrap(coords, in);
	for(int i=0;i<n*n*n*3;i++)
		printf("c = %f\n",coords[i] );
	//rand_wrapper(0);
	//rand_wrapper(0);
	return 0;
}*/

/*void gpu_NN_start(float coords_list, float * NNlist, int numC) // wrapper to initiate kernels
{
	// coords_list needs to be 1D list of coords
	// allocate host and device memory
	// copy coords list to gpu
	// set up grid and blocks
	// execute kernel
	// copy nnlist back
	// free memory
	// exit
}*/